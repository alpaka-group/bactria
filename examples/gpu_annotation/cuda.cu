#include "hip/hip_runtime.h"
/* Copyright (c) 2014, NVIDIA CORPORATION. All rights reserved.
 * Copyright (c) 2021, Jan Stephan
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <bactria/bactria.hpp>

#include <cmath>
#include <cstddef>
#include <cstdlib>
#include <iostream>
#include <stdexcept>

__global__ void init_data_kernel(int n, double* x)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n)
    {
        x[i] = n - i;
    }
}

__global__ void daxpy_kernel(int n, double a, double* x, double* y)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n)
    {
        y[i] = a * x[i] + y[i];
    }
}

__global__ void check_results_kernel(int n, double correctvalue, double* x)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n)
    {
        if(x[i] != correctvalue)
        {
            printf("ERROR at index = %d, expected = %f, actual: %f\n", i, correctvalue, x[i]);
        }
    }
}

auto init_host_data(std::size_t n, double* x)
{
    bactria::ranges::Range{"init_host_data", bactria::ranges::color::blue};
    for(auto i = std::size_t{0}; i < n; ++i)
    {
        x[i] = i;
    }
}

auto init_data(std::size_t n, double* x, double* x_d, double* y_d)
{
    bactria::ranges::Range{"init_host_data", bactria::ranges::color::yellow};
    hipStream_t copy_stream;
    hipStream_t compute_stream;
    hipStreamCreate(&copy_stream);
    hipStreamCreate(&compute_stream);

    hipMemcpyAsync(x_d, x, n * sizeof(double), hipMemcpyDefault, copy_stream);
    init_data_kernel<<<static_cast<int>(std::ceil(n / 256.f)), 256, 0, compute_stream>>>(n, y_d);

    hipStreamSynchronize(copy_stream);
    hipStreamSynchronize(compute_stream);

    hipStreamDestroy(compute_stream);
    hipStreamDestroy(copy_stream);
}

auto daxpy(std::size_t n, double a, double* x_d, double* y_d)
{
    bactria::ranges::Range{"daxpy", bactria::ranges::color::magenta};
    daxpy_kernel<<<static_cast<int>(std::ceil(n / 256.f)), 256>>>(n, a, x_d, y_d);
    hipDeviceSynchronize();
}

auto check_results(std::size_t n, double correctvalue, double* x_d)
{
    bactria::ranges::Range{"check_results", bactria::ranges::color::cyan};
    check_results_kernel<<<static_cast<int>(std::ceil(n / 256.f)), 256>>>(n, correctvalue, x_d);
}

auto run_test(std::size_t n)
{
    bactria::ranges::Range{"run_test", bactria::ranges::color::lime};
    double* x;
    double* x_d;
    double* y_d;
    hipSetDevice(0);
    hipHostMalloc((void**) &x, n * sizeof(double));
    hipMalloc((void**) &x_d, n * sizeof(double));
    hipMalloc((void**) &y_d, n * sizeof(double));

    init_host_data(n, x);

    init_data(n, x, x_d, y_d);

    daxpy(n, 1.0, x_d, y_d);

    check_results(n, n, y_d);

    hipFree(y_d);
    hipFree(x_d);
    hipHostFree(x);
    hipDeviceSynchronize();
}

auto main() -> int
{
    try
    {
        auto ctx = bactria::Context{};
        constexpr auto n = std::size_t{1} << 22;
        run_test(n);
    }
    catch(std::runtime_error const& err)
    {
        std::cerr << err.what() << std::endl;
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}